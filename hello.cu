#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "timer.h"

__global__ void vectorAdd(float *a,float *b, float *c, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N){
        c[i] = a[i] + b[i];
    }
}


int main(){

    hipDeviceSynchronize();

    const int N = 10000000;


    float *a = new float[N];
    float *b = new float[N];
    float *out = new float[N];
    float *out_p = new float[N];

    for (int i = 0; i < N; i++){
        a[i] = rand();
        b[i] = rand();
    }

    const size_t Size = N * sizeof(float);

    core::timer cpu_t;
    cpu_t.start();
    for (int i = 0; i < N; i++){
        out[i] = a[i] + b[i];
    }
    printf("Cpu time taken :- %f ns\n",cpu_t.nanoseconds());

    float *d_a, *d_b, *d_out;
    hipMalloc(&d_a, Size);
    hipMalloc(&d_b, Size);
    hipMalloc(&d_out, Size);

    hipMemcpy(d_a,a,Size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,Size,hipMemcpyHostToDevice);

    //still don't understand this part :(
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    core::timer gpu_t;
    gpu_t.start();

    vectorAdd<<<blocksPerGrid,threadsPerBlock>>>(d_a,d_b,d_out,N);

    printf("gpu time taken :- %f ns\n",gpu_t.nanoseconds());

    hipMemcpy(d_out,out_p,Size,hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    return 0;
}